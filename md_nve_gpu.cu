
/*  ----------------------------------------------------------
	Project name  : Molecular Dynamics Simulation on GPU
	Description   : This program simulates many particles 
					system using CUDA thechnology on
					cuda enabled nvidia GPU.
	
	What have done:
	- A simple (loop over all particles) & 2D code implemented
    ---------------------------------------------------------- */ 

#include "hip/hip_runtime.h"


// Includes

#include<cstdio>
#include<iostream>
#include<fstream>
using namespace std;


// Macroes and Symbolic Constants

#define N		10000 // number of particles
#define d       	2.5f // lattice distance
#define L		(sqrtf(N)*d) // simulation box length
#define h		.0001f // time step
#define vmax		10.f // maximum random velocity
#define block_dim	32  // number of threads in each block
#define RAND		(2.f*((float)rand()/RAND_MAX-.5f))

// Typedefs

typedef struct
{
	float2 r;
	float2 v;
	float2 f;
} Particle;

// Functions

__host__   void setVCMzero(Particle *);
__global__ void kernel_ComputeForce(void *);
__host__   void InitializeParticle(Particle *);
__global__ void kernel_TimeIntegration_0(void *);
__global__ void kernel_TimeIntegration_1(void *);
__device__ float2 TowParticlesInteraction(float2, float2, float2);

// Implementation of functions

int main()
{
	// Multiple GPUs --------------------------
	int n_GPU;
	hipGetDeviceCount(&n_GPU); 
	hipDeviceProp_t prop;

	std::cout<<"Number of GPU is "<<n_GPU<<std::endl;
	for(int i=0;i<n_GPU;++i)
	{
		
		hipGetDeviceProperties(&prop,i);
		std::cout<<"GPU No. "<<i<<" "<< prop.name <<std::endl;
	}
	hipSetDevice(0);
	std::cout<<std::endl;

	Particle *dev_P; // device particles pointer
	Particle *hst_P = new Particle[N]; // host particles pointer allocation
	const int size = N*sizeof(Particle); // size of allocated memory
	ofstream out("configuration.xyz");
	InitializeParticle(hst_P);
	hipMalloc((void**)&dev_P, size); // device particle pointer allocation
	hipMemcpy(dev_P, hst_P, size, hipMemcpyHostToDevice); // host to device memory copy
	kernel_ComputeForce<<<N/block_dim, block_dim>>>(dev_P); // compute force kernel function
	for(int t=0;t<=10000;t++)
	{
		if(t%100==0)
		{
			hipMemcpy(hst_P, dev_P, size, hipMemcpyDeviceToHost); // device to host memory copy
    			cout << " t --> " << t << endl;
			out << N << "\n\n";
			for(int i=0;i<N;i++)
				out <<"Ar\t"<<(hst_P+i)->r.x<<"\t"<<(hst_P+i)->r.y<<"\t"<<0.f<<endl;
		}
		// fast verlet time integration
		kernel_TimeIntegration_0<<<N/block_dim, block_dim>>>(dev_P);
		kernel_ComputeForce<<<N/block_dim, block_dim>>>(dev_P);
		kernel_TimeIntegration_1<<<N/block_dim, block_dim>>>(dev_P);
		
	}
	out.close();
	delete[] hst_P;
	hipFree(dev_P);
	return 0;
}

__host__ void InitializeParticle(Particle *P)
{
	for (int j=0, k=0;j<(int)sqrt((double)N);j++) 
		for (int i=0;i<(int)sqrt((double)N);i++,k++) 
		{
			// initialize particle's coordinates
			(P+k)->r.x = (float)i*d; 
			(P+k)->r.y = (float)j*d; 	
			// initialize velocities
			(P+k)->v.x = vmax*RAND;
			(P+k)->v.y = vmax*RAND;
		}
	setVCMzero(P);
}

__host__ void setVCMzero(Particle *P)
{   
	int i;
	float2	Vcm={0.f, 0.f};
	for(i=0;i<N;i++) {
		Vcm.x+=(P+i)->v.x;
		Vcm.y+=(P+i)->v.y;
	}
	for(i=0;i<N;i++) {
		(P+i)->v.x-=Vcm.x/N;
		(P+i)->v.y-=Vcm.y/N;
	}
}
		
__device__ float2 TowParticlesInteraction(float2 ri, float2 rj, float2 fi)
{	
	float r;
	float2 rij, fij; 
    // periodic boundary condtion
	rij.x = rj.x - ri.x;
	if (rij.x>=L/2.f) 
		rij.x-=L;
	else if (rij.x<=-L/2.f) 
			rij.x+=L;

	rij.y = rj.y - ri.y;
	if (rij.y>=L/2.f) 
		rij.y-=L;
	else if (rij.y<=-L/2.f)
			rij.y+=L;	
	// LJ force field
	r = sqrtf(rij.x*rij.x + rij.y*rij.y);	
	if (r>0.000001f)
	{
		r*=r;
		fij.x = rij.x/r; 
		fij.y = rij.y/r;
		r*= r*r;
		fij.x*= (6.f/r-12.f/r/r); 
		fij.y*= (6.f/r-12.f/r/r);
		// update forces
		fi.x += fij.x;
		fi.y += fij.y;
	}
	return fi;
}

__global__ void kernel_ComputeForce(void *global_P)
{
	
	float2 force = {0.f, 0.f};
	Particle *P =(Particle *)global_P;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// compute force for each thread
	for (int i=0;i<N;i++)
		if(i != idx)
			force = TowParticlesInteraction((P+idx)->r, (P+i)->r, force);
	(P+idx)->f = force;
	__syncthreads();
}

__global__ void kernel_TimeIntegration_0(void *global_P)
{
	Particle *P =(Particle *)global_P;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// update coordinates and velocities
	(P+idx)->r.x=fmod((P+idx)->r.x + h * ( (P+idx)->v.x + 0.5f*h*(P+idx)->f.x ) + 10.f*L, L);
	(P+idx)->r.y=fmod((P+idx)->r.y + h * ( (P+idx)->v.y + 0.5f*h*(P+idx)->f.y ) + 10.f*L, L);
	(P+idx)->v.x+=0.5f*h*(P+idx)->f.x;
	(P+idx)->v.y+=0.5f*h*(P+idx)->f.y;	
	__syncthreads();
}

__global__ void kernel_TimeIntegration_1(void *global_P)
{
	Particle *P =(Particle *)global_P;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// ubdate velocities with new force
	(P+idx)->v.x+= 0.5f*h*(P+idx)->f.x;
	(P+idx)->v.y+= 0.5f*h*(P+idx)->f.y;			
	__syncthreads();
}

